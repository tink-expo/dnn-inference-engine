
#include <hip/hip_runtime.h>
#include "string.h"
#include "stdio.h"
#include "stdlib.h"
#include "math.h"

#define THREADS_2D 16
#define THREADS_1D 256

#define MAX(x, y) (x >= y ? x : y)

__global__ void h_cuda_im2col(float* im_b, float* col_b,
        int oh, int ow,
        int iw, int ic,
        int kh, int kw, 
        int sh, int sw)
{
    int col_w = ic * kh * kw;
    int col_i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (col_i_idx < oh * ow) {
        int patch_i = (col_i_idx / ow) * sh;
        int patch_j = (col_i_idx % ow) * sw;
        for (int c = 0; c < ic; ++c) {
            int col_j = c * (kh * kw);
            for (int di = 0; di < kh; ++di) {
                for (int dj = 0; dj < kw; ++dj) {
                    col_b[col_i_idx * col_w +
                            col_j + (di * kw) + dj] = 
                            im_b[(patch_i + di) * (iw * ic) +
                            (patch_j + dj) * ic +
                            c];
                }
            }
        }
    }
}

__global__ void h_cuda_matmul(float* imcol, float* kernel, float* result, 
        int m_size, int n_size, int k_size)
{
    int i_idx = blockIdx.y * blockDim.y + threadIdx.y;
    int j_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx < m_size && j_idx < n_size) {
        float res = 0.0f;
        for (int k = 0; k < k_size; ++k) {
            res += imcol[i_idx * k_size + k] * kernel[k * n_size + j_idx];
        }
        result[i_idx * n_size + j_idx] = res;
    }
}

__global__ void h_cuda_batch_norm(float* alpha, float* beta, float* result,
        int r_size, int od)
{
    int t_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (t_idx < r_size) {
        int d = t_idx % od;
        result[t_idx] = result[t_idx] * alpha[d] - beta[d];
    }
}

__global__ void h_cuda_batch_norm2(float* in_layer, float* alpha, float* beta, float* result,
    int r_size, int od)
{
    int t_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (t_idx < r_size) {
        int d = t_idx % od;
        result[t_idx] = in_layer[t_idx] * alpha[d] - beta[d];
    }
}

__global__ void h_cuda_max_pool2d(
        float* in_layer, float* result,
        int r_size, 
        int oh, int ow, int od, 
        int ih, int iw, int ic,
        int kh, int kw,
        int sh, int sw)
{
    int t_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (t_idx < r_size) {
        // Calc i, j, d.
        int d = t_idx;
        int i = d / (ow * od);
        d -= i * (ow * od);
        int j = d / od;
        d -= j * od;
        
        int ii = (i * sh) * (iw * ic) + (j * sw) * ic + d;
        
        float imax = in_layer[ii];
        for (int di = 0; di < kh; ++di) {
            for (int dj = 0; dj < kw; ++dj) {
                if (di > 0 || dj > 0) {
                    imax = MAX(imax, 
                            in_layer[ii + di * (iw * ic) + dj * ic]);
                }
            }
        }
        result[t_idx] = imax;
    }
}

// void im2col(float* im_b_arg,
//     float* col_b_arg,
//     int oh, int ow,
//     int ih, int iw, int ic,
//     int kh, int kw, 
//     int sh, int sw)
// {
//     float (*im_b)[iw][ic] = (float (*)[iw][ic]) im_b_arg;
//     float (*col_b)[ic * kh * kw] = (float (*)[ic * kh * kw]) col_b_arg;

//     for (int i = 0; i < oh; ++i) {
//         for (int j = 0; j < ow; ++j) {
//             int patch_i = i * sh;
//             int patch_j = j * sw;
//             for (int c = 0; c < ic; ++c) {
//                 int col_i = i * ow + j;
//                 int col_j = c * (kh * kw);
//                 for (int di = 0; di < kh; ++di) {
//                     for (int dj = 0; dj < kw; ++dj) {
//                         col_b[col_i][col_j + (di * kw) + dj] = 
//                                 im_b[patch_i + di][patch_j + dj][c];
//                     }
//                 }
//             }
//         }
//     }
// }

void im2col(float* im_b,
        float* col_b,
        int oh, int ow,
        int ih, int iw, int ic,
        int kh, int kw, 
        int sh, int sw)
{
    // float (*im_b)[iw][ic] = (float (*)[iw][ic]) im_b_arg;
    // float (*col_b)[ic * kh * kw] = (float (*)[ic * kh * kw]) col_b_arg;

    int col_w = ic * kh * kw;
    for (int i = 0; i < oh; ++i) {
        for (int j = 0; j < ow; ++j) {
            int patch_i = i * sh;
            int patch_j = j * sw;
            for (int c = 0; c < ic; ++c) {
                int col_i = i * ow + j;
                int col_j = c * (kh * kw);
                for (int di = 0; di < kh; ++di) {
                    for (int dj = 0; dj < kw; ++dj) {
                        col_b[col_i * col_w +
                                col_j + (di * kw) + dj] = 
                                im_b[(patch_i + di) * (iw * ic) +
                                (patch_j + dj) * ic +
                                c];
                    }
                }
            }
        }
    }
}

extern "C" {

void conv2d_cuda(float* in_layer,
        float* col,
        float* kernel_r, 
        float* result,
        int batch, int oh, int ow, int od,
        int ih, int iw, int ic,
        int kh, int kw,
        int sh, int sw)
{
    for (int b = 0; b < batch; ++b) {
        float* im_b = in_layer + b * (ih * iw * ic);
        float* col_b = col + b * ((oh * ow) * (ic * kh * kw));
        float* result_b = result + b * (oh * ow * od);

        im2col(im_b,
                col_b,
                oh, ow,
                ih, iw, ic,
                kh, kw,
                sh, sw);

        // col_b : (oh * ow) X (ic * kh * kw)
        // kernel_r : (ic * kh * kw) X od

        int m_size = oh * ow;
        int n_size = od;
        int k_size = ic * kh * kw;

        float* d_imcol;
        float* d_kernel;
        float* d_result;
        hipMalloc((void **) &d_imcol, sizeof(float) * m_size * k_size);
        hipMalloc((void **) &d_kernel, sizeof(float) * k_size * n_size);
        hipMalloc((void **) &d_result, sizeof(float) * m_size * k_size);

        hipMemcpy(d_imcol, col_b, sizeof(float) * m_size * k_size, hipMemcpyHostToDevice);
        hipMemcpy(d_kernel, kernel_r, sizeof(float) * k_size * n_size, hipMemcpyHostToDevice);
        
        // TODO: Optimize here for Yolov2tiny size
        unsigned int grid_r = (m_size + THREADS_2D - 1) / THREADS_2D;
        unsigned int grid_c = (n_size + THREADS_2D - 1) / THREADS_2D;
        dim3 grid_dim(grid_c, grid_r);
        dim3 block_dim(THREADS_2D, THREADS_2D);

        h_cuda_matmul<<<grid_dim, block_dim>>>(d_imcol, d_kernel, d_result, m_size, n_size, k_size);
        hipFree(d_imcol);
        hipFree(d_kernel);

        hipMemcpy(result_b, d_result, sizeof(float) * m_size * n_size, hipMemcpyDeviceToHost);
        hipFree(d_result);
    }
}


void conv2d_cuda_im2col_cuda(float* in_layer,
    float* col,
    float* kernel_r, 
    float* result,
    int batch, int oh, int ow, int od,
    int ih, int iw, int ic,
    int kh, int kw,
    int sh, int sw)
{
for (int b = 0; b < batch; ++b) {
    float* im_b = in_layer + b * (ih * iw * ic);
    float* col_b = col + b * ((oh * ow) * (ic * kh * kw));
    float* result_b = result + b * (oh * ow * od);

    im2col(im_b,
            col_b,
            oh, ow,
            ih, iw, ic,
            kh, kw,
            sh, sw);

    // col_b : (oh * ow) X (ic * kh * kw)
    // kernel_r : (ic * kh * kw) X od
    
    int im_size = ih * iw * ic;
    int m_size = oh * ow;
    int n_size = od;
    int k_size = ic * kh * kw;
    
    float* d_im;
    float* d_col;
    float* d_kernel;
    float* d_result;
    hipMalloc((void **) &d_im, sizeof(float) * im_size);
    hipMalloc((void **) &d_col, sizeof(float) * m_size * k_size);
    hipMemcpy(d_im, im_b, sizeof(float) * im_size, hipMemcpyHostToDevice);

    unsigned int grid_m = (m_size + THREADS_1D - 1) / THREADS_1D;
    dim3 grid_m_dim(grid_m);
    dim3 block_m_dim(THREADS_1D);

    h_cuda_im2col<<<grid_m_dim, block_m_dim>>>(d_im, d_col,
            oh, ow, iw, ic, kh, kw, sh, sw);
    hipFree(d_im);

    hipMalloc((void **) &d_kernel, sizeof(float) * k_size * n_size);
    hipMalloc((void **) &d_result, sizeof(float) * m_size * k_size);
    hipMemcpy(d_kernel, kernel_r, sizeof(float) * k_size * n_size, hipMemcpyHostToDevice);
    
    // TODO: Optimize here for Yolov2tiny size
    unsigned int grid_r = (m_size + THREADS_2D - 1) / THREADS_2D;
    unsigned int grid_c = (n_size + THREADS_2D - 1) / THREADS_2D;
    dim3 grid_dim(grid_c, grid_r);
    dim3 block_dim(THREADS_2D, THREADS_2D);

    h_cuda_matmul<<<grid_dim, block_dim>>>(d_col, d_kernel, d_result, m_size, n_size, k_size);
    hipFree(d_col);
    hipFree(d_kernel);

    hipMemcpy(result_b, d_result, sizeof(float) * m_size * n_size, hipMemcpyDeviceToHost);
    hipFree(d_result);
}
}


void bias_add(float* in_layer, float* biases, float* result,
    int batch, int oh, int ow, int od) 
{
    for (int b = 0; b < batch; ++b) {
        for (int i = 0; i < oh; ++i) {
            for (int j = 0; j < ow; ++j) {
                for (int d = 0; d < od; ++d) {
                    int ri = b * (oh * ow * od) +
                            i * (ow * od) +
                            j * od +
                            d;
                    result[ri] = in_layer[ri] + biases[d];
                }
            }
        }
    }
}

void batch_norm_cuda(float* in_layer,
    float* alpha,
    float* beta,
    float* result,
    int batch, int oh, int ow, int od)
{
    int r_size = batch * oh * ow * od;
    memcpy(result, in_layer, sizeof(float) * r_size); 

    float* d_alpha;
    float* d_beta;
    float* d_result;

    hipMalloc((void **) &d_alpha, sizeof(float) * od);
    hipMalloc((void **) &d_beta, sizeof(float) * od);
    hipMalloc((void **) &d_result, sizeof(float) * r_size);

    hipMemcpy(d_result, result, sizeof(float) * r_size, hipMemcpyHostToDevice);
    hipMemcpy(d_alpha, alpha, sizeof(float) * od, hipMemcpyHostToDevice);
    hipMemcpy(d_beta, beta, sizeof(float) * od, hipMemcpyHostToDevice);
    
    unsigned int grid_size = (r_size + THREADS_1D - 1) / THREADS_1D;
    dim3 grid_dim(grid_size);
    dim3 block_dim(THREADS_1D);

    h_cuda_batch_norm<<<grid_dim, block_dim>>>(d_alpha, d_beta, d_result, r_size, od);
    hipFree(d_alpha);
    hipFree(d_beta);

    hipMemcpy(result, d_result, sizeof(float) * r_size, hipMemcpyDeviceToHost);
    hipFree(d_result);
}

void batch_norm_cuda2(float* in_layer,
    float* alpha,
    float* beta,
    float* result,
    int batch, int oh, int ow, int od)
{
    int r_size = batch * oh * ow * od;

    float* d_in_layer;
    float* d_alpha;
    float* d_beta;
    float* d_result;

    hipMalloc((void **) &d_in_layer, sizeof(float) * r_size);
    hipMalloc((void **) &d_alpha, sizeof(float) * od);
    hipMalloc((void **) &d_beta, sizeof(float) * od);
    hipMalloc((void **) &d_result, sizeof(float) * r_size);

    hipMemcpy(d_in_layer, in_layer, sizeof(float) * r_size, hipMemcpyHostToDevice);
    hipMemcpy(d_alpha, alpha, sizeof(float) * od, hipMemcpyHostToDevice);
    hipMemcpy(d_beta, beta, sizeof(float) * od, hipMemcpyHostToDevice);
    
    unsigned int grid_size = (r_size + THREADS_1D - 1) / THREADS_1D;
    dim3 grid_dim(grid_size);
    dim3 block_dim(THREADS_1D);

    h_cuda_batch_norm2<<<grid_dim, block_dim>>>(d_in_layer, d_alpha, d_beta, d_result, r_size, od);
    hipFree(d_in_layer);
    hipFree(d_alpha);
    hipFree(d_beta);

    hipMemcpy(result, d_result, sizeof(float) * r_size, hipMemcpyDeviceToHost);
    hipFree(d_result);
}



void max_pool2d(float* in_layer,
        float* result,
        int batch, int oh, int ow, int od,
        int ih, int iw, int ic,
        int kh, int kw,
        int sh, int sw)
{
    for (int b = 0; b < batch; ++b) {
        for (int i = 0; i < oh; ++i) {
            for (int j = 0; j < ow; ++j) {
                for (int d = 0; d < od; ++d) {
                    int ii = (i * sh) * (iw * ic) + (j * sw) * ic + d;
                    float imax = in_layer[ii];
                    for (int di = 0; di < kh; ++di) {
                        for (int dj = 0; dj < kw; ++dj) {
                            if (di > 0 || dj > 0) {
                                imax = MAX(imax, 
                                        in_layer[ii + di * (iw * ic) + dj * ic]);
                            }
                        }
                    }
                    result[
                            b * (oh * ow * od) +
                            i * (ow * od) +
                            j * od +
                            d
                    ] = imax;
                }
            }
        }
    }
}

void max_pool2d_test(float* in_layer,
    float* result,
    int batch, int oh, int ow, int od,
    int ih, int iw, int ic,
    int kh, int kw,
    int sh, int sw)
{
    for (int b = 0; b < batch; ++b) {
        int r_size = oh * ow * od;

        float* in_layer_b = in_layer + b * (ih * iw * ic);
        float* result_b = result + b * (oh * ow * od);

        for (int t_idx = 0; t_idx < r_size; ++t_idx) {
            int d = t_idx;
            int i = d / (ow * od);
            d -= i * (ow * od);
            int j = d / od;
            d -= j * od;

            int ii = (i * sh) * (iw * ic) + (j * sw) * ic + d;
            float imax = in_layer_b[ii];
            for (int di = 0; di < kh; ++di) {
                for (int dj = 0; dj < kw; ++dj) {
                    if (di > 0 || dj > 0) {
                        imax = MAX(imax, 
                                in_layer_b[ii + di * (iw * ic) + dj * ic]);
                    }
                }
            }
            result_b[t_idx] = imax;
        }
    }
}

void max_pool2d_cuda(float* in_layer,
    float* result,
    int batch, int oh, int ow, int od,
    int ih, int iw, int ic,
    int kh, int kw,
    int sh, int sw)
{
    for (int b = 0; b < batch; ++b) {
        int r_size = oh * ow * od;
        int i_size = ih * iw * ic;

        float* in_layer_b = in_layer + b * (ih * iw * ic);
        float* result_b = result + b * (oh * ow * od);
        
        float* d_in_layer;
        float* d_result;

        hipMalloc((void **) &d_in_layer, sizeof(float) * i_size);
        hipMalloc((void **) &d_result, sizeof(float) * r_size);

        hipMemcpy(d_in_layer, in_layer_b, sizeof(float) * i_size, hipMemcpyHostToDevice);

        unsigned int grid_size = (r_size + THREADS_1D - 1) / THREADS_1D;
        dim3 grid_dim(grid_size);
        dim3 block_dim(THREADS_1D);

        h_cuda_max_pool2d<<<grid_dim, block_dim>>>(
                d_in_layer, d_result, 
                r_size,
                oh, ow, od, 
                ih, iw, ic, 
                kh, kw,
                sh, sw);
        hipFree(d_in_layer);

        hipMemcpy(result_b, d_result, sizeof(float) * r_size, hipMemcpyDeviceToHost);
        hipFree(d_result);
    }
}

void leaky_relu(float* in_layer,
        float* result,
        int batch, int oh, int ow, int od)
{
    for (int b = 0; b < batch; ++b) {
        for (int i = 0; i < oh; ++i) {
            for (int j = 0; j < ow; ++j) {
                for (int d = 0; d < od; ++d) {
                    int idx = b * (oh * ow * od) +
                            i * (ow * od) +
                            j * od +
                            d;
                    float t = in_layer[idx];
                    result[idx] = t < 0 ? 0.1 * t : t;
                }
            }
        }
    }
}
}