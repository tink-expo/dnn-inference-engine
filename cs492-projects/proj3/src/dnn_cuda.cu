
#include <hip/hip_runtime.h>
#include "string.h"
#include "stdio.h"
#include "stdlib.h"
#include "math.h"
#include "pthread.h"

#define CUDA_THREADS_2D 16
#define CUDA_THREADS_1D 256

#define MAX(x, y) (x >= y ? x : y)

#define P_THREADS 4

__global__ void h_cuda_im2col(float* im_b, float* col_b,
        int oh, int ow,
        int iw, int ic,
        int kh, int kw, 
        int sh, int sw)
{
    int col_w = ic * kh * kw;
    int col_i_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (col_i_idx < oh * ow) {
        int patch_i = (col_i_idx / ow) * sh;
        int patch_j = (col_i_idx % ow) * sw;
        for (int c = 0; c < ic; ++c) {
            int col_j = c * (kh * kw);
            for (int di = 0; di < kh; ++di) {
                for (int dj = 0; dj < kw; ++dj) {
                    col_b[col_i_idx * col_w +
                            col_j + (di * kw) + dj] = 
                            im_b[(patch_i + di) * (iw * ic) +
                            (patch_j + dj) * ic +
                            c];
                }
            }
        }
    }
}

__global__ void h_cuda_matmul(float* imcol, float* kernel, float* result, 
        int m_size, int n_size, int k_size)
{
    int i_idx = blockIdx.y * blockDim.y + threadIdx.y;
    int j_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_idx < m_size && j_idx < n_size) {
        float res = 0.0f;
        for (int k = 0; k < k_size; ++k) {
            res += imcol[i_idx * k_size + k] * kernel[k * n_size + j_idx];
        }
        result[i_idx * n_size + j_idx] = res;
    }
}

__global__ void h_cuda_batch_norm(float* alpha, float* beta, float* result,
        int r_size, int od)
{
    int t_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (t_idx < r_size) {
        int d = t_idx % od;
        result[t_idx] = result[t_idx] * alpha[d] - beta[d];
    }
}

__global__ void h_cuda_batch_norm2(float* in_layer, float* alpha, float* beta, float* result,
    int r_size, int od)
{
    int t_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (t_idx < r_size) {
        int d = t_idx % od;
        result[t_idx] = in_layer[t_idx] * alpha[d] - beta[d];
    }
}

__global__ void h_cuda_max_pool2d(
        float* in_layer, float* result,
        int r_size, 
        int oh, int ow, int od, 
        int ih, int iw, int ic,
        int kh, int kw,
        int sh, int sw)
{
    int t_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (t_idx < r_size) {
        // Calc i, j, d.
        int d = t_idx;
        int i = d / (ow * od);
        d -= i * (ow * od);
        int j = d / od;
        d -= j * od;
        
        int ii = (i * sh) * (iw * ic) + (j * sw) * ic + d;
        
        float imax = in_layer[ii];
        for (int di = 0; di < kh; ++di) {
            for (int dj = 0; dj < kw; ++dj) {
                if (di > 0 || dj > 0) {
                    imax = MAX(imax, 
                            in_layer[ii + di * (iw * ic) + dj * ic]);
                }
            }
        }
        result[t_idx] = imax;
    }
}

void im2col(float* im_b,
        float* col_b,
        int oh, int ow,
        int ih, int iw, int ic,
        int kh, int kw, 
        int sh, int sw)
{
    int col_w = ic * kh * kw;
    for (int i = 0; i < oh; ++i) {
        for (int j = 0; j < ow; ++j) {
            int patch_i = i * sh;
            int patch_j = j * sw;
            for (int c = 0; c < ic; ++c) {
                int col_i = i * ow + j;
                int col_j = c * (kh * kw);
                for (int di = 0; di < kh; ++di) {
                    for (int dj = 0; dj < kw; ++dj) {
                        col_b[col_i * col_w +
                                col_j + (di * kw) + dj] = 
                                im_b[(patch_i + di) * (iw * ic) +
                                (patch_j + dj) * ic +
                                c];
                    }
                }
            }
        }
    }
}

struct shape_arg {
    int batch, oh, ow, od;
    int ih, iw, ic;
    int kh, kw;
    int sh, sw;
};

struct im2col_thread_arg {
    float* im_b;
    float* col_b;
    struct shape_arg* shape;
    int oh_s;
    int oh_e;
};

void* im2col_thread_func(void* thread_arg)
{
    struct im2col_thread_arg* arg = (struct im2col_thread_arg*) thread_arg;
    struct shape_arg* shape = arg->shape;

    int col_w = shape->ic * shape->kh * shape->kw;
    for (int i = arg->oh_s; i < arg->oh_e; ++i) {
        for (int j = 0; j < shape->ow; ++j) {
            int patch_i = i * shape->sh;
            int patch_j = j * shape->sw;
            for (int c = 0; c < shape->ic; ++c) {
                int col_i = i * shape->ow + j;
                int col_j = c * (shape->kh * shape->kw);
                for (int di = 0; di < shape->kh; ++di) {
                    for (int dj = 0; dj < shape->kw; ++dj) {
                        arg->col_b[col_i * col_w +
                                col_j + (di * shape->kw) + dj] = 
                                arg->im_b[(patch_i + di) * (shape->iw * shape->ic) +
                                (patch_j + dj) * shape->ic +
                                c];
                    }
                }
            }
        }
    }

    return 0;
}

extern "C" {

void conv2d_cuda_pthread(float* in_layer,
        float* col,
        float* kernel_r, 
        float* result,
        int* shape_arg_arr)
{
    struct shape_arg* shape = (struct shape_arg*) shape_arg_arr;
    
    for (int b = 0; b < shape->batch; ++b) {
        float* im_b = in_layer + b * (shape->ih * shape->iw * shape->ic);
        float* col_b = col + b * ((shape->oh * shape->ow) * (shape->ic * shape->kh * shape->kw));
        float* result_b = result + b * (shape->oh * shape->ow * shape->od);

        pthread_t threads[P_THREADS];
        struct im2col_thread_arg t_args[P_THREADS];
        int oh_part_size = shape->oh / P_THREADS;
        

        t_args[0].im_b = im_b;
        t_args[0].col_b = col_b;
        t_args[0].shape = shape;

        int t_id;

        for (int t_idx = 0; t_idx < P_THREADS; ++t_idx) {
            if (t_idx > 0) {
                t_args[t_idx] = t_args[0];
            }

            int oh_s = oh_part_size * t_idx;
            int oh_e = t_idx < P_THREADS - 1 ? oh_s + oh_part_size : shape->oh;
            
            t_args[t_idx].oh_s = oh_s;
            t_args[t_idx].oh_e = oh_e;

            t_id = pthread_create(&threads[t_idx], NULL, im2col_thread_func, (void*) &t_args[t_idx]);
            if (t_id < 0) {
                perror("conv2d im2col thread error : ");
                exit(0);
            }
        }

        for (int t_idx = 0; t_idx < P_THREADS; ++t_idx) {
            pthread_join(threads[t_idx], NULL);
        }

        // col_b : (oh * ow) X (ic * kh * kw)
        // kernel_r : (ic * kh * kw) X od

        int m_size = shape->oh * shape->ow;
        int n_size = shape->od;
        int k_size = shape->ic * shape->kh * shape->kw;

        float* d_imcol;
        float* d_kernel;
        float* d_result;
        hipMalloc((void **) &d_imcol, sizeof(float) * m_size * k_size);
        hipMalloc((void **) &d_kernel, sizeof(float) * k_size * n_size);
        hipMalloc((void **) &d_result, sizeof(float) * m_size * k_size);

        hipMemcpy(d_imcol, col_b, sizeof(float) * m_size * k_size, hipMemcpyHostToDevice);
        hipMemcpy(d_kernel, kernel_r, sizeof(float) * k_size * n_size, hipMemcpyHostToDevice);
        
        // TODO: Optimize here for Yolov2tiny size
        unsigned int grid_r = (m_size + CUDA_THREADS_2D - 1) / CUDA_THREADS_2D;
        unsigned int grid_c = (n_size + CUDA_THREADS_2D - 1) / CUDA_THREADS_2D;
        dim3 grid_dim(grid_c, grid_r);
        dim3 block_dim(CUDA_THREADS_2D, CUDA_THREADS_2D);

        h_cuda_matmul<<<grid_dim, block_dim>>>(d_imcol, d_kernel, d_result, m_size, n_size, k_size);
        hipFree(d_imcol);
        hipFree(d_kernel);

        hipMemcpy(result_b, d_result, sizeof(float) * m_size * n_size, hipMemcpyDeviceToHost);
        hipFree(d_result);
    }
}

void conv2d_cuda(float* in_layer,
        float* col,
        float* kernel_r, 
        float* result,
        int batch, int oh, int ow, int od,
        int ih, int iw, int ic,
        int kh, int kw,
        int sh, int sw)
{
    for (int b = 0; b < batch; ++b) {
        float* im_b = in_layer + b * (ih * iw * ic);
        float* col_b = col + b * ((oh * ow) * (ic * kh * kw));
        float* result_b = result + b * (oh * ow * od);

        im2col(im_b,
                col_b,
                oh, ow,
                ih, iw, ic,
                kh, kw,
                sh, sw);

        // col_b : (oh * ow) X (ic * kh * kw)
        // kernel_r : (ic * kh * kw) X od

        int m_size = oh * ow;
        int n_size = od;
        int k_size = ic * kh * kw;

        float* d_imcol;
        float* d_kernel;
        float* d_result;
        hipMalloc((void **) &d_imcol, sizeof(float) * m_size * k_size);
        hipMalloc((void **) &d_kernel, sizeof(float) * k_size * n_size);
        hipMalloc((void **) &d_result, sizeof(float) * m_size * k_size);

        hipMemcpy(d_imcol, col_b, sizeof(float) * m_size * k_size, hipMemcpyHostToDevice);
        hipMemcpy(d_kernel, kernel_r, sizeof(float) * k_size * n_size, hipMemcpyHostToDevice);
        
        // TODO: Optimize here for Yolov2tiny size
        unsigned int grid_r = (m_size + CUDA_THREADS_2D - 1) / CUDA_THREADS_2D;
        unsigned int grid_c = (n_size + CUDA_THREADS_2D - 1) / CUDA_THREADS_2D;
        dim3 grid_dim(grid_c, grid_r);
        dim3 block_dim(CUDA_THREADS_2D, CUDA_THREADS_2D);

        h_cuda_matmul<<<grid_dim, block_dim>>>(d_imcol, d_kernel, d_result, m_size, n_size, k_size);
        hipFree(d_imcol);
        hipFree(d_kernel);

        hipMemcpy(result_b, d_result, sizeof(float) * m_size * n_size, hipMemcpyDeviceToHost);
        hipFree(d_result);
    }
}


void conv2d_cuda_im2col_cuda(float* in_layer,
    float* col,
    float* kernel_r, 
    float* result,
    int batch, int oh, int ow, int od,
    int ih, int iw, int ic,
    int kh, int kw,
    int sh, int sw)
{
for (int b = 0; b < batch; ++b) {
    float* im_b = in_layer + b * (ih * iw * ic);
    float* col_b = col + b * ((oh * ow) * (ic * kh * kw));
    float* result_b = result + b * (oh * ow * od);

    im2col(im_b,
            col_b,
            oh, ow,
            ih, iw, ic,
            kh, kw,
            sh, sw);

    // col_b : (oh * ow) X (ic * kh * kw)
    // kernel_r : (ic * kh * kw) X od
    
    int im_size = ih * iw * ic;
    int m_size = oh * ow;
    int n_size = od;
    int k_size = ic * kh * kw;
    
    float* d_im;
    float* d_col;
    float* d_kernel;
    float* d_result;
    hipMalloc((void **) &d_im, sizeof(float) * im_size);
    hipMalloc((void **) &d_col, sizeof(float) * m_size * k_size);
    hipMemcpy(d_im, im_b, sizeof(float) * im_size, hipMemcpyHostToDevice);

    unsigned int grid_m = (m_size + CUDA_THREADS_1D - 1) / CUDA_THREADS_1D;
    dim3 grid_m_dim(grid_m);
    dim3 block_m_dim(CUDA_THREADS_1D);

    h_cuda_im2col<<<grid_m_dim, block_m_dim>>>(d_im, d_col,
            oh, ow, iw, ic, kh, kw, sh, sw);
    hipFree(d_im);

    hipMalloc((void **) &d_kernel, sizeof(float) * k_size * n_size);
    hipMalloc((void **) &d_result, sizeof(float) * m_size * k_size);
    hipMemcpy(d_kernel, kernel_r, sizeof(float) * k_size * n_size, hipMemcpyHostToDevice);
    
    // TODO: Optimize here for Yolov2tiny size
    unsigned int grid_r = (m_size + CUDA_THREADS_2D - 1) / CUDA_THREADS_2D;
    unsigned int grid_c = (n_size + CUDA_THREADS_2D - 1) / CUDA_THREADS_2D;
    dim3 grid_dim(grid_c, grid_r);
    dim3 block_dim(CUDA_THREADS_2D, CUDA_THREADS_2D);

    h_cuda_matmul<<<grid_dim, block_dim>>>(d_col, d_kernel, d_result, m_size, n_size, k_size);
    hipFree(d_col);
    hipFree(d_kernel);

    hipMemcpy(result_b, d_result, sizeof(float) * m_size * n_size, hipMemcpyDeviceToHost);
    hipFree(d_result);
}
}


void bias_add(float* in_layer, float* biases, float* result,
    int batch, int oh, int ow, int od) 
{
    for (int b = 0; b < batch; ++b) {
        for (int i = 0; i < oh; ++i) {
            for (int j = 0; j < ow; ++j) {
                for (int d = 0; d < od; ++d) {
                    int ri = b * (oh * ow * od) +
                            i * (ow * od) +
                            j * od +
                            d;
                    result[ri] = in_layer[ri] + biases[d];
                }
            }
        }
    }
}

void batch_norm_cuda(float* in_layer,
    float* alpha,
    float* beta,
    float* result,
    int batch, int oh, int ow, int od)
{
    int r_size = batch * oh * ow * od;
    memcpy(result, in_layer, sizeof(float) * r_size); 

    float* d_alpha;
    float* d_beta;
    float* d_result;

    hipMalloc((void **) &d_alpha, sizeof(float) * od);
    hipMalloc((void **) &d_beta, sizeof(float) * od);
    hipMalloc((void **) &d_result, sizeof(float) * r_size);

    hipMemcpy(d_result, result, sizeof(float) * r_size, hipMemcpyHostToDevice);
    hipMemcpy(d_alpha, alpha, sizeof(float) * od, hipMemcpyHostToDevice);
    hipMemcpy(d_beta, beta, sizeof(float) * od, hipMemcpyHostToDevice);
    
    unsigned int grid_size = (r_size + CUDA_THREADS_1D - 1) / CUDA_THREADS_1D;
    dim3 grid_dim(grid_size);
    dim3 block_dim(CUDA_THREADS_1D);

    h_cuda_batch_norm<<<grid_dim, block_dim>>>(d_alpha, d_beta, d_result, r_size, od);
    hipFree(d_alpha);
    hipFree(d_beta);

    hipMemcpy(result, d_result, sizeof(float) * r_size, hipMemcpyDeviceToHost);
    hipFree(d_result);
}

void batch_norm_cuda2(float* in_layer,
    float* alpha,
    float* beta,
    float* result,
    int batch, int oh, int ow, int od)
{
    int r_size = batch * oh * ow * od;

    float* d_in_layer;
    float* d_alpha;
    float* d_beta;
    float* d_result;

    hipMalloc((void **) &d_in_layer, sizeof(float) * r_size);
    hipMalloc((void **) &d_alpha, sizeof(float) * od);
    hipMalloc((void **) &d_beta, sizeof(float) * od);
    hipMalloc((void **) &d_result, sizeof(float) * r_size);

    hipMemcpy(d_in_layer, in_layer, sizeof(float) * r_size, hipMemcpyHostToDevice);
    hipMemcpy(d_alpha, alpha, sizeof(float) * od, hipMemcpyHostToDevice);
    hipMemcpy(d_beta, beta, sizeof(float) * od, hipMemcpyHostToDevice);
    
    unsigned int grid_size = (r_size + CUDA_THREADS_1D - 1) / CUDA_THREADS_1D;
    dim3 grid_dim(grid_size);
    dim3 block_dim(CUDA_THREADS_1D);

    h_cuda_batch_norm2<<<grid_dim, block_dim>>>(d_in_layer, d_alpha, d_beta, d_result, r_size, od);
    hipFree(d_in_layer);
    hipFree(d_alpha);
    hipFree(d_beta);

    hipMemcpy(result, d_result, sizeof(float) * r_size, hipMemcpyDeviceToHost);
    hipFree(d_result);
}



void max_pool2d(float* in_layer,
        float* result,
        int batch, int oh, int ow, int od,
        int ih, int iw, int ic,
        int kh, int kw,
        int sh, int sw)
{
    for (int b = 0; b < batch; ++b) {
        for (int i = 0; i < oh; ++i) {
            for (int j = 0; j < ow; ++j) {
                for (int d = 0; d < od; ++d) {
                    int ii = (i * sh) * (iw * ic) + (j * sw) * ic + d;
                    float imax = in_layer[ii];
                    for (int di = 0; di < kh; ++di) {
                        for (int dj = 0; dj < kw; ++dj) {
                            if (di > 0 || dj > 0) {
                                imax = MAX(imax, 
                                        in_layer[ii + di * (iw * ic) + dj * ic]);
                            }
                        }
                    }
                    result[
                            b * (oh * ow * od) +
                            i * (ow * od) +
                            j * od +
                            d
                    ] = imax;
                }
            }
        }
    }
}

void max_pool2d_test(float* in_layer,
    float* result,
    int batch, int oh, int ow, int od,
    int ih, int iw, int ic,
    int kh, int kw,
    int sh, int sw)
{
    for (int b = 0; b < batch; ++b) {
        int r_size = oh * ow * od;

        float* in_layer_b = in_layer + b * (ih * iw * ic);
        float* result_b = result + b * (oh * ow * od);

        for (int t_idx = 0; t_idx < r_size; ++t_idx) {
            int d = t_idx;
            int i = d / (ow * od);
            d -= i * (ow * od);
            int j = d / od;
            d -= j * od;

            int ii = (i * sh) * (iw * ic) + (j * sw) * ic + d;
            float imax = in_layer_b[ii];
            for (int di = 0; di < kh; ++di) {
                for (int dj = 0; dj < kw; ++dj) {
                    if (di > 0 || dj > 0) {
                        imax = MAX(imax, 
                                in_layer_b[ii + di * (iw * ic) + dj * ic]);
                    }
                }
            }
            result_b[t_idx] = imax;
        }
    }
}

void max_pool2d_cuda(float* in_layer,
    float* result,
    int batch, int oh, int ow, int od,
    int ih, int iw, int ic,
    int kh, int kw,
    int sh, int sw)
{
    for (int b = 0; b < batch; ++b) {
        int r_size = oh * ow * od;
        int i_size = ih * iw * ic;

        float* in_layer_b = in_layer + b * (ih * iw * ic);
        float* result_b = result + b * (oh * ow * od);
        
        float* d_in_layer;
        float* d_result;

        hipMalloc((void **) &d_in_layer, sizeof(float) * i_size);
        hipMalloc((void **) &d_result, sizeof(float) * r_size);

        hipMemcpy(d_in_layer, in_layer_b, sizeof(float) * i_size, hipMemcpyHostToDevice);

        unsigned int grid_size = (r_size + CUDA_THREADS_1D - 1) / CUDA_THREADS_1D;
        dim3 grid_dim(grid_size);
        dim3 block_dim(CUDA_THREADS_1D);

        h_cuda_max_pool2d<<<grid_dim, block_dim>>>(
                d_in_layer, d_result, 
                r_size,
                oh, ow, od, 
                ih, iw, ic, 
                kh, kw,
                sh, sw);
        hipFree(d_in_layer);

        hipMemcpy(result_b, d_result, sizeof(float) * r_size, hipMemcpyDeviceToHost);
        hipFree(d_result);
    }
}

void leaky_relu(float* in_layer,
        float* result,
        int batch, int oh, int ow, int od)
{
    for (int b = 0; b < batch; ++b) {
        for (int i = 0; i < oh; ++i) {
            for (int j = 0; j < ow; ++j) {
                for (int d = 0; d < od; ++d) {
                    int idx = b * (oh * ow * od) +
                            i * (ow * od) +
                            j * od +
                            d;
                    float t = in_layer[idx];
                    result[idx] = t < 0 ? 0.1 * t : t;
                }
            }
        }
    }
}
}